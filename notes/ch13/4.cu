#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

// Reference CPU implementation using std::sort for verification
void cpu_reference_sort(int *input, int *output, int N) {
  // Copy input to output first
  for (int i = 0; i < N; i++) {
    output[i] = input[i];
  }

  // Simple sort using STL
  std::sort(output, output + N);
}

// Function to verify results between CPU and GPU implementations
bool verify_results(int *cpu_results, int *gpu_results, int N) {
  for (int i = 0; i < N; i++) {
    if (cpu_results[i] != gpu_results[i]) {
      printf("Mismatch at index %d: CPU = %d, GPU = %d\n", i, cpu_results[i],
             gpu_results[i]);
      return false;
    }
  }
  return true;
}

// A simplified GPU sort that just uses thrust sort for validation
void gpu_reference_sort(int *d_input, int *d_output, int N) {
  // Copy d_input to d_output
  hipMemcpy(d_output, d_input, N * sizeof(int), hipMemcpyDeviceToDevice);

  // For now, we'll do the sorting on CPU and copy back
  int *h_temp = new int[N];
  hipMemcpy(h_temp, d_input, N * sizeof(int), hipMemcpyDeviceToHost);
  std::sort(h_temp, h_temp + N);
  hipMemcpy(d_output, h_temp, N * sizeof(int), hipMemcpyHostToDevice);
  delete[] h_temp;
}

int main() {
  const int N = 1000000; // 1M elements

  // Allocate host memory
  int *h_input = new int[N];
  int *h_output = new int[N];
  int *h_cpu_output = new int[N];

  // Initialize input data with random values - using fixed seed for consistent
  // results
  srand(12345);
  for (int i = 0; i < N; i++) {
    h_input[i] = rand() % 1000000;
  }

  // Allocate device memory
  int *d_input, *d_output;
  hipMalloc((void **)&d_input, N * sizeof(int));
  hipMalloc((void **)&d_output, N * sizeof(int));

  // Copy input data to device
  hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ---- CPU Reference Implementation ----
  auto cpu_start = std::chrono::high_resolution_clock::now();
  cpu_reference_sort(h_input, h_cpu_output, N);
  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<float, std::milli> cpu_duration = cpu_end - cpu_start;

  // ---- GPU Reference Implementation ----
  hipEventRecord(start);
  gpu_reference_sort(d_input, d_output, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float gpu_duration = 0.0f;
  hipEventElapsedTime(&gpu_duration, start, stop);

  // Copy results back to host
  hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

  // Verify results
  bool results_match = verify_results(h_cpu_output, h_output, N);

  // Print timing information and verification results
  printf("Validating CPU and GPU reference implementations on %d elements\n",
         N);
  printf("CPU Time: %.2f ms\n", cpu_duration.count());
  printf("GPU Time: %.2f ms\n", gpu_duration);
  printf("Speedup: %.2fx\n", cpu_duration.count() / gpu_duration);
  printf("Verification: %s\n", results_match ? "PASSED" : "FAILED");

  // Free memory
  delete[] h_input;
  delete[] h_output;
  delete[] h_cpu_output;
  hipFree(d_input);
  hipFree(d_output);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}