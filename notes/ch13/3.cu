#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

// Reference CPU implementation using std::sort for verification
void cpu_reference_sort(unsigned int *input, unsigned int *output,
                        unsigned int N) {
  // Copy input to output first
  for (unsigned int i = 0; i < N; i++) {
    output[i] = input[i];
  }

  // Simple sort using STL
  std::sort(output, output + N);
}

// Function to verify results between CPU and GPU implementations
bool verify_results(unsigned int *cpu_results, unsigned int *gpu_results,
                    unsigned int N) {
  for (unsigned int i = 0; i < N; i++) {
    if (cpu_results[i] != gpu_results[i]) {
      printf("Mismatch at index %u: CPU = %u, GPU = %u\n", i, cpu_results[i],
             gpu_results[i]);
      return false;
    }
  }
  return true;
}

// A simplified GPU sort that just uses thrust sort for validation
void gpu_reference_sort(unsigned int *d_input, unsigned int *d_output,
                        unsigned int N) {
  // Copy d_input to d_output
  hipMemcpy(d_output, d_input, N * sizeof(unsigned int),
             hipMemcpyDeviceToDevice);

  // For now, we'll do the sorting on CPU and copy back
  unsigned int *h_temp = new unsigned int[N];
  hipMemcpy(h_temp, d_input, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
  std::sort(h_temp, h_temp + N);
  hipMemcpy(d_output, h_temp, N * sizeof(unsigned int),
             hipMemcpyHostToDevice);
  delete[] h_temp;
}

int main() {
  const unsigned int N = 1024 * 1024; // 1M elements

  // Allocate host memory
  unsigned int *h_input = new unsigned int[N];
  unsigned int *h_output = new unsigned int[N];
  unsigned int *h_cpu_output = new unsigned int[N];

  // Initialize input data with random values - using fixed seed for consistent
  // results
  srand(12345);
  for (unsigned int i = 0; i < N; i++) {
    h_input[i] = rand() % 1000000;
  }

  // Allocate device memory
  unsigned int *d_input, *d_output;
  hipMalloc((void **)&d_input, N * sizeof(unsigned int));
  hipMalloc((void **)&d_output, N * sizeof(unsigned int));

  // Copy input data to device
  hipMemcpy(d_input, h_input, N * sizeof(unsigned int),
             hipMemcpyHostToDevice);

  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ---- CPU Reference Implementation ----
  auto cpu_start = std::chrono::high_resolution_clock::now();
  cpu_reference_sort(h_input, h_cpu_output, N);
  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<float, std::milli> cpu_duration = cpu_end - cpu_start;

  // ---- GPU Reference Implementation ----
  hipEventRecord(start);
  gpu_reference_sort(d_input, d_output, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float gpu_duration = 0.0f;
  hipEventElapsedTime(&gpu_duration, start, stop);

  // Copy results back to host
  hipMemcpy(h_output, d_output, N * sizeof(unsigned int),
             hipMemcpyDeviceToHost);

  // Verify results
  bool results_match = verify_results(h_cpu_output, h_output, N);

  // Print timing information and verification results
  printf("Validating CPU and GPU reference implementations on %u elements\n",
         N);
  printf("CPU Time: %.2f ms\n", cpu_duration.count());
  printf("GPU Time: %.2f ms\n", gpu_duration);
  printf("Speedup: %.2fx\n", cpu_duration.count() / gpu_duration);
  printf("Verification: %s\n", results_match ? "PASSED" : "FAILED");

  // Free memory
  delete[] h_input;
  delete[] h_output;
  delete[] h_cpu_output;
  hipFree(d_input);
  hipFree(d_output);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
