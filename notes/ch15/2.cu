#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define INF -1   // Used to mark unvisited vertices
#define ALPHA 14 // Constant for switching from push to pull
#define BETA 24  // Constant for switching from pull to push

struct Graph {
  int num_vertices;
  int num_edges;
  int *row_ptr;
  int *col_idx;
};

struct BFSData {
  int *dist;
  int *visited;  // Changed from bool to int to avoid alignment issues
  int *frontier;
  int *new_frontier;
  int *frontier_size;
  int *new_frontier_size;
};

void initGraph(Graph &graph, int num_vertices, int num_edges, int *row_ptr,
               int *col_idx) {
  graph.num_vertices = num_vertices;
  graph.num_edges = num_edges;

  hipMalloc((void **)&graph.row_ptr, (num_vertices + 1) * sizeof(int));
  hipMemcpy(graph.row_ptr, row_ptr, (num_vertices + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&graph.col_idx, num_edges * sizeof(int));
  hipMemcpy(graph.col_idx, col_idx, num_edges * sizeof(int),
             hipMemcpyHostToDevice);
}

void initBFSData(BFSData &bfs_data, int num_vertices, int source) {
  int *h_dist = (int *)malloc(num_vertices * sizeof(int));
  int *h_visited = (int *)malloc(num_vertices * sizeof(int));  // Changed from bool to int

  for (int i = 0; i < num_vertices; i++) {
    h_dist[i] = INF;
    h_visited[i] = 0;  // 0 = false, 1 = true
  }
  h_dist[source] = 0;
  h_visited[source] = 1;  // Mark source as visited

  hipMalloc((void **)&bfs_data.dist, num_vertices * sizeof(int));
  hipMalloc((void **)&bfs_data.visited, num_vertices * sizeof(int));  // Changed from bool to int
  hipMalloc((void **)&bfs_data.frontier, num_vertices * sizeof(int));
  hipMalloc((void **)&bfs_data.new_frontier, num_vertices * sizeof(int));
  hipMalloc((void **)&bfs_data.frontier_size, sizeof(int));
  hipMalloc((void **)&bfs_data.new_frontier_size, sizeof(int));

  hipMemcpy(bfs_data.dist, h_dist, num_vertices * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(bfs_data.visited, h_visited, num_vertices * sizeof(int),
             hipMemcpyHostToDevice);

  int initial_frontier[1] = {source};
  int initial_size = 1;
  hipMemcpy(bfs_data.frontier, initial_frontier, sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(bfs_data.frontier_size, &initial_size, sizeof(int),
             hipMemcpyHostToDevice);

  int zero = 0;
  hipMemcpy(bfs_data.new_frontier_size, &zero, sizeof(int),
             hipMemcpyHostToDevice);

  free(h_dist);
  free(h_visited);
}

void cleanupBFSData(BFSData &bfs_data) {
  hipFree(bfs_data.dist);
  hipFree(bfs_data.visited);
  hipFree(bfs_data.frontier);
  hipFree(bfs_data.new_frontier);
  hipFree(bfs_data.frontier_size);
  hipFree(bfs_data.new_frontier_size);
}

void cleanupGraph(Graph &graph) {
  hipFree(graph.row_ptr);
  hipFree(graph.col_idx);
}

__global__ void bfsPushKernel(Graph graph, BFSData bfs_data, int level) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < *bfs_data.frontier_size) {
    int vertex = bfs_data.frontier[tid];

    int start = graph.row_ptr[vertex];
    int end = graph.row_ptr[vertex + 1];

    for (int edge = start; edge < end; edge++) {
      int neighbor = graph.col_idx[edge];

      // Change from bool to int comparison
      if (bfs_data.visited[neighbor] == 0) {
        // Compare and swap with int values instead of bool
        int old_val = atomicCAS(&bfs_data.visited[neighbor], 0, 1);
        if (old_val == 0) {
          int index = atomicAdd(bfs_data.new_frontier_size, 1);
          bfs_data.new_frontier[index] = neighbor;
          bfs_data.dist[neighbor] = level + 1;
        }
      }
    }
  }
}

__global__ void bfsPullKernel(Graph graph, BFSData bfs_data, int level) {
  int vertex = blockIdx.x * blockDim.x + threadIdx.x;

  if (vertex < graph.num_vertices && bfs_data.dist[vertex] == INF) {
    for (int i = 0; i < graph.num_vertices; i++) {
      // Check if vertex i is at the current level and has an edge to vertex
      if (bfs_data.dist[i] == level) {
        int start = graph.row_ptr[i];
        int end = graph.row_ptr[i + 1];

        for (int edge = start; edge < end; edge++) {
          if (graph.col_idx[edge] == vertex) {
            bfs_data.dist[vertex] = level + 1;
            bfs_data.visited[vertex] = 1;  // Changed from bool to int
            int index = atomicAdd(bfs_data.new_frontier_size, 1);
            bfs_data.new_frontier[index] = vertex;
            return;
          }
        }
      }
    }
  }
}

bool shouldUsePull(int frontier_size, int num_edges, int num_vertices,
                   int edges_examined_push) {
  return (frontier_size * ALPHA > num_edges);
}

bool shouldUsePush(int frontier_size, int num_vertices) {
  return (frontier_size * BETA < num_vertices);
}

void directionOptimizedBFS(Graph &graph, BFSData &bfs_data, int source) {
  int level = 0;
  int h_frontier_size = 1; // Start with just the source vertex
  bool using_pull = false;

  int block_size = 256;
  while (h_frontier_size > 0) {
    if (!using_pull && shouldUsePull(h_frontier_size, graph.num_edges,
                                     graph.num_vertices, 0)) {
      using_pull = true;
      printf("Switching to PULL at level %d, frontier size: %d\n", level,
             h_frontier_size);
    } else if (using_pull &&
               shouldUsePush(h_frontier_size, graph.num_vertices)) {
      using_pull = false;
      printf("Switching to PUSH at level %d, frontier size: %d\n", level,
             h_frontier_size);
    }

    int zero = 0;
    hipMemcpy(bfs_data.new_frontier_size, &zero, sizeof(int),
               hipMemcpyHostToDevice);

    if (using_pull) {
      int grid_size = (graph.num_vertices + block_size - 1) / block_size;
      bfsPullKernel<<<grid_size, block_size>>>(graph, bfs_data, level);
    } else {
      int grid_size = (h_frontier_size + block_size - 1) / block_size;
      bfsPushKernel<<<grid_size, block_size>>>(graph, bfs_data, level);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      break;
    }

    hipDeviceSynchronize();

    int *temp_frontier = bfs_data.frontier;
    bfs_data.frontier = bfs_data.new_frontier;
    bfs_data.new_frontier = temp_frontier;

    hipMemcpy(&h_frontier_size, bfs_data.new_frontier_size, sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(bfs_data.frontier_size, &h_frontier_size, sizeof(int),
               hipMemcpyHostToDevice);

    level++;
  }

  printf("BFS completed after %d levels\n", level);
}

int main(int argc, char **argv) {
  int num_vertices = 8;
  int num_edges = 15;

  int row_ptr[9] = {0, 2, 5, 6, 8, 9, 11, 12, 15};
  int col_idx[15] = {2, 5, 0, 4, 7, 3, 0, 6, 3, 1, 7, 4, 2, 4, 6};

  Graph graph;
  initGraph(graph, num_vertices, num_edges, row_ptr, col_idx);

  BFSData bfs_data;
  int source = 0;
  initBFSData(bfs_data, num_vertices, source);

  directionOptimizedBFS(graph, bfs_data, source);

  int *h_dist = (int *)malloc(num_vertices * sizeof(int));
  hipMemcpy(h_dist, bfs_data.dist, num_vertices * sizeof(int),
             hipMemcpyDeviceToHost);

  printf("Distances from source vertex %d:\n", source);
  for (int i = 0; i < num_vertices; i++) {
    if (h_dist[i] != INF) {
      printf("Vertex %d: Distance %d\n", i, h_dist[i]);
    } else {
      printf("Vertex %d: Unreachable\n", i);
    }
  }

  free(h_dist);
  cleanupBFSData(bfs_data);
  cleanupGraph(graph);

  return 0;
}