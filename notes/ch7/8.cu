#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>

#define TILE_WIDTH 16

__global__ void convolution_3D_basic_kernel(float *N, float *F, float *P, int r,
                                            int width, int height, int depth) {
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;
  int outZ = blockIdx.z;

  if (outCol >= width || outRow >= height || outZ >= depth)
    return;

  float Pvalue = 0.0f;

  for (int fZ = 0; fZ < 2 * r + 1; fZ++) {
    for (int fRow = 0; fRow < 2 * r + 1; fRow++) {
      for (int fCol = 0; fCol < 2 * r + 1; fCol++) {
        int inZ = outZ - r + fZ;
        int inRow = outRow - r + fRow;
        int inCol = outCol - r + fCol;

        if (inZ >= 0 && inZ < depth && inRow >= 0 && inRow < height &&
            inCol >= 0 && inCol < width) {

          int filterIdx =
              fZ * (2 * r + 1) * (2 * r + 1) + fRow * (2 * r + 1) + fCol;
          int inputIdx = inZ * height * width + inRow * width + inCol;

          Pvalue += F[filterIdx] * N[inputIdx];
        }
      }
    }
  }

  int outputIdx = outZ * height * width + outRow * width + outCol;
  P[outputIdx] = Pvalue;
}

// CPU implementation of 3D convolution
void convolution_3D_cpu(float *N, float *F, float *P, int r, int width, int height, int depth) {
  for (int outZ = 0; outZ < depth; outZ++) {
    for (int outRow = 0; outRow < height; outRow++) {
      for (int outCol = 0; outCol < width; outCol++) {
        float Pvalue = 0.0f;
        
        for (int fZ = 0; fZ < 2 * r + 1; fZ++) {
          for (int fRow = 0; fRow < 2 * r + 1; fRow++) {
            for (int fCol = 0; fCol < 2 * r + 1; fCol++) {
              int inZ = outZ - r + fZ;
              int inRow = outRow - r + fRow;
              int inCol = outCol - r + fCol;
              
              if (inZ >= 0 && inZ < depth && inRow >= 0 && inRow < height &&
                  inCol >= 0 && inCol < width) {
                
                int filterIdx = fZ * (2 * r + 1) * (2 * r + 1) + fRow * (2 * r + 1) + fCol;
                int inputIdx = inZ * height * width + inRow * width + inCol;
                
                Pvalue += F[filterIdx] * N[inputIdx];
              }
            }
          }
        }
        
        int outputIdx = outZ * height * width + outRow * width + outCol;
        P[outputIdx] = Pvalue;
      }
    }
  }
}

int main() {
  unsigned int channels = 3;
  unsigned int filter_size = 3;
  unsigned int r = filter_size / 2;
  unsigned int dim = 64;

  size_t volume_size = channels * dim * dim * sizeof(float);
  float *N = static_cast<float *>(malloc(volume_size));
  float *P = static_cast<float *>(malloc(volume_size));
  float *P_cpu = static_cast<float *>(malloc(volume_size));

  size_t filter_cube_size =
      (2 * r + 1) * (2 * r + 1) * (2 * r + 1) * sizeof(float);
  float *F = static_cast<float *>(malloc(filter_cube_size));

  // Initialize input and filter with some values
  for (unsigned int i = 0; i < channels * dim * dim; i++) {
    N[i] = 1.0f;
  }

  for (unsigned int i = 0; i < (2 * r + 1) * (2 * r + 1) * (2 * r + 1); i++) {
    F[i] = 1.0f / ((2 * r + 1) * (2 * r + 1) * (2 * r + 1));
  }

  // CPU timing with CUDA events
  hipEvent_t start_cpu, stop_cpu;
  hipEventCreate(&start_cpu);
  hipEventCreate(&stop_cpu);
  
  hipEventRecord(start_cpu);
  convolution_3D_cpu(N, F, P_cpu, r, dim, dim, channels);
  hipEventRecord(stop_cpu);
  hipEventSynchronize(stop_cpu);
  
  float cpu_time = 0.0f;
  hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);
  printf("CPU execution time: %f milliseconds\n", cpu_time);

  // GPU timing with CUDA events
  hipEvent_t start_gpu, stop_gpu, start_kernel, stop_kernel;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  
  hipEventRecord(start_gpu);
  
  float *N_d = nullptr, *F_d = nullptr, *P_d = nullptr;

  hipMalloc(reinterpret_cast<void **>(&N_d), volume_size);
  hipMalloc(reinterpret_cast<void **>(&F_d), filter_cube_size);
  hipMalloc(reinterpret_cast<void **>(&P_d), volume_size);

  hipMemcpy(N_d, N, volume_size, hipMemcpyHostToDevice);
  hipMemcpy(F_d, F, filter_cube_size, hipMemcpyHostToDevice);

  dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
  dim3 gridSize((dim + blockSize.x - 1) / blockSize.x,
                (dim + blockSize.y - 1) / blockSize.y, channels);

  hipEventRecord(start_kernel);
  convolution_3D_basic_kernel<<<gridSize, blockSize>>>(N_d, F_d, P_d, r, dim,
                                                       dim, channels);
  hipEventRecord(stop_kernel);
  hipEventSynchronize(stop_kernel);
  
  float kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);
  printf("GPU kernel execution time: %f milliseconds\n", kernel_time);
  
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Copy the result back to host
  hipMemcpy(P, P_d, volume_size, hipMemcpyDeviceToHost);

  hipFree(N_d);
  hipFree(F_d);
  hipFree(P_d);

  hipEventRecord(stop_gpu);
  hipEventSynchronize(stop_gpu);
  
  float gpu_time = 0.0f;
  hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);
  printf("GPU total execution time: %f milliseconds\n", gpu_time);
  
  // Verify results - calculate max error and mean error
  float max_error = 0.0f;
  float mean_error = 0.0f;
  int error_count = 0;
  
  for (unsigned int i = 0; i < channels * dim * dim; i++) {
    float error = fabs(P[i] - P_cpu[i]);
    if (error > max_error) {
      max_error = error;
    }
    if (error > 1e-3) { // Count significant errors
      error_count++;
    }
    mean_error += error;
  }
  mean_error /= (channels * dim * dim);
  
  // Report verification results
  printf("Verification results:\n");
  printf("  Max error: %e\n", max_error);
  printf("  Mean error: %e\n", mean_error);
  printf("  Error count (>1e-3): %d out of %d elements\n", error_count, channels * dim * dim);
  
  // Determine if verification passed using a reasonable threshold
  bool verification_passed = (max_error < 1e-2) && (error_count < channels * dim * dim / 1000);
  printf("%s\n", verification_passed ? "Verification PASSED!" : "Verification FAILED!");
  
  // Calculate speedup
  printf("Speedup (CPU vs GPU kernel): %.2fx\n", cpu_time / kernel_time);
  printf("Speedup (CPU vs GPU total): %.2fx\n", cpu_time / gpu_time);
  
  // Cleanup events
  hipEventDestroy(start_cpu);
  hipEventDestroy(stop_cpu);
  hipEventDestroy(start_gpu);
  hipEventDestroy(stop_gpu);
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);

  // Free memory
  free(N);
  free(F);
  free(P);
  free(P_cpu);

  return 0;
}