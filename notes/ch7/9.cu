#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>

#define TILE_WIDTH 16
#define FILTER_SIZE 3
#define RADIUS (FILTER_SIZE / 2)

__constant__ float F_d[2 * RADIUS + 1][2 * RADIUS + 1][2 * RADIUS + 1];

__global__ void convolution_3D_basic_kernel(float *N, float *F, float *P,
                                            int width, int height, int depth) {
  int outCol = blockIdx.x * blockDim.x + threadIdx.x;
  int outRow = blockIdx.y * blockDim.y + threadIdx.y;
  int outZ = blockIdx.z;

  if (outCol >= width || outRow >= height || outZ >= depth)
    return;

  float Pvalue = 0.0f;

  for (int fZ = 0; fZ < FILTER_SIZE; fZ++) {
    for (int fRow = 0; fRow < FILTER_SIZE; fRow++) {
      for (int fCol = 0; fCol < FILTER_SIZE; fCol++) {
        int inZ = outZ - RADIUS + fZ;
        int inRow = outRow - RADIUS + fRow;
        int inCol = outCol - RADIUS + fCol;

        if (inZ >= 0 && inZ < depth && inRow >= 0 && inRow < height &&
            inCol >= 0 && inCol < width) {

          int inputIdx = inZ * height * width + inRow * width + inCol;

          Pvalue += F_d[fZ][fRow][fCol] * N[inputIdx];
        }
      }
    }
  }

  int outputIdx = outZ * height * width + outRow * width + outCol;
  P[outputIdx] = Pvalue;
}

// CPU implementation of 3D convolution
void convolution_3D_cpu(float *N, float *F, float *P, int width, int height, int depth) {
  for (int outZ = 0; outZ < depth; outZ++) {
    for (int outRow = 0; outRow < height; outRow++) {
      for (int outCol = 0; outCol < width; outCol++) {
        float Pvalue = 0.0f;
        
        for (int fZ = 0; fZ < FILTER_SIZE; fZ++) {
          for (int fRow = 0; fRow < FILTER_SIZE; fRow++) {
            for (int fCol = 0; fCol < FILTER_SIZE; fCol++) {
              int inZ = outZ - RADIUS + fZ;
              int inRow = outRow - RADIUS + fRow;
              int inCol = outCol - RADIUS + fCol;
              
              if (inZ >= 0 && inZ < depth && inRow >= 0 && inRow < height &&
                  inCol >= 0 && inCol < width) {
                
                int filterIdx = fZ * FILTER_SIZE * FILTER_SIZE + fRow * FILTER_SIZE + fCol;
                int inputIdx = inZ * height * width + inRow * width + inCol;
                
                Pvalue += F[filterIdx] * N[inputIdx];
              }
            }
          }
        }
        
        int outputIdx = outZ * height * width + outRow * width + outCol;
        P[outputIdx] = Pvalue;
      }
    }
  }
}

int main() {
  unsigned int channels = 3;
  unsigned int dim = 64;

  size_t volume_size = channels * dim * dim * sizeof(float);
  float *N = static_cast<float *>(malloc(volume_size));
  float *P = static_cast<float *>(malloc(volume_size));
  float *P_cpu = static_cast<float *>(malloc(volume_size));

  float *F = static_cast<float *>(
      malloc(FILTER_SIZE * FILTER_SIZE * FILTER_SIZE * sizeof(float)));

  // Initialize input and filter with some values
  for (unsigned int i = 0; i < channels * dim * dim; i++) {
    N[i] = 1.0f;
  }

  for (unsigned int i = 0; i < FILTER_SIZE * FILTER_SIZE * FILTER_SIZE; i++) {
    F[i] = 1.0f / (FILTER_SIZE * FILTER_SIZE * FILTER_SIZE);
  }

  // CPU timing with CUDA events
  hipEvent_t start_cpu, stop_cpu;
  hipEventCreate(&start_cpu);
  hipEventCreate(&stop_cpu);
  
  hipEventRecord(start_cpu);
  convolution_3D_cpu(N, F, P_cpu, dim, dim, channels);
  hipEventRecord(stop_cpu);
  hipEventSynchronize(stop_cpu);
  
  float cpu_time = 0.0f;
  hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);
  printf("CPU execution time: %f milliseconds\n", cpu_time);

  // GPU timing with CUDA events
  hipEvent_t start_gpu, stop_gpu, start_kernel, stop_kernel;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  
  hipEventRecord(start_gpu);
  float *N_d = nullptr, *P_d = nullptr;

  hipMemcpyToSymbol(HIP_SYMBOL(F_d), F,
                     FILTER_SIZE * FILTER_SIZE * FILTER_SIZE * sizeof(float));

  hipMalloc(reinterpret_cast<void **>(&N_d), volume_size);
  hipMalloc(reinterpret_cast<void **>(&P_d), volume_size);

  hipMemcpy(N_d, N, volume_size, hipMemcpyHostToDevice);

  dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
  dim3 gridSize((dim + blockSize.x - 1) / blockSize.x,
                (dim + blockSize.y - 1) / blockSize.y, channels);

  hipEventRecord(start_kernel);
  convolution_3D_basic_kernel<<<gridSize, blockSize>>>(N_d, nullptr, P_d, dim, dim,
                                                       channels);
  hipEventRecord(stop_kernel);
  hipEventSynchronize(stop_kernel);
  
  float kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);
  printf("GPU kernel execution time: %f milliseconds\n", kernel_time);
  
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Copy the result back to host
  hipMemcpy(P, P_d, volume_size, hipMemcpyDeviceToHost);

  hipFree(N_d);
  hipFree(P_d);

  hipEventRecord(stop_gpu);
  hipEventSynchronize(stop_gpu);
  
  float gpu_time = 0.0f;
  hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);
  printf("GPU total execution time: %f milliseconds\n", gpu_time);
  
  // Verify results - calculate max error and mean error
  float max_error = 0.0f;
  float mean_error = 0.0f;
  int error_count = 0;
  
  for (unsigned int i = 0; i < channels * dim * dim; i++) {
    float error = fabs(P[i] - P_cpu[i]);
    if (error > max_error) {
      max_error = error;
    }
    if (error > 1e-3) { // Count significant errors
      error_count++;
    }
    mean_error += error;
  }
  mean_error /= (channels * dim * dim);
  
  // Report verification results
  printf("Verification results:\n");
  printf("  Max error: %e\n", max_error);
  printf("  Mean error: %e\n", mean_error);
  printf("  Error count (>1e-3): %d out of %d elements\n", error_count, channels * dim * dim);
  
  // Determine if verification passed using a reasonable threshold
  bool verification_passed = (max_error < 1e-2) && (error_count < channels * dim * dim / 1000);
  printf("%s\n", verification_passed ? "Verification PASSED!" : "Verification FAILED!");
  
  // Calculate speedup
  printf("Speedup (CPU vs GPU kernel): %.2fx\n", cpu_time / kernel_time);
  printf("Speedup (CPU vs GPU total): %.2fx\n", cpu_time / gpu_time);
  
  // Cleanup events
  hipEventDestroy(start_cpu);
  hipEventDestroy(stop_cpu);
  hipEventDestroy(start_gpu);
  hipEventDestroy(stop_gpu);
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);

  // Free memory
  free(N);
  free(F);
  free(P);
  free(P_cpu);

  return 0;
}