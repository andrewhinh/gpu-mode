#include "hip/hip_runtime.h"
#include "../common.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cmath>

__host__ __device__ float matmul_col(float *M, float *N, int row, int col,
                                     int width) {
  float pVal = 0.0;
  for (int k = 0; k < width; ++k) {
    pVal += M[row * width + k] * N[k * width + col];
  }
  return pVal;
}

__global__ void gpu(float *M, float *N, float *P, int width) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col < width) {
    for (int row = 0; row < width; ++row) {
      P[row * width + col] = matmul_col(M, N, row, col, width);
    }
  }
}

__host__ void cpu(float *M, float *N, float *P, int width) {
  for (int col = 0; col < width; ++col) {
    for (int row = 0; row < width; ++row) {
      P[row * width + col] = matmul_col(M, N, row, col, width);
    }
  }
}

int main() {
  int width = 1024;
  float *M, *N, *P;
  M = static_cast<float *>(malloc(width * width * sizeof(float)));
  N = static_cast<float *>(malloc(width * width * sizeof(float)));
  P = static_cast<float *>(malloc(width * width * sizeof(float)));

  // Initialize matrices with small random values to prevent overflow
  srand(42); // Use fixed seed for reproducibility
  for (int i = 0; i < width * width; ++i) {
    M[i] = static_cast<float>(rand() % 10) / 10.0f;
    N[i] = static_cast<float>(rand() % 10) / 10.0f;
  }

  // CPU timing with CUDA events
  hipEvent_t start_cpu, stop_cpu;
  hipEventCreate(&start_cpu);
  hipEventCreate(&stop_cpu);
  
  hipEventRecord(start_cpu);
  cpu(M, N, P, width);
  hipEventRecord(stop_cpu);
  hipEventSynchronize(stop_cpu);
  
  float cpu_time = 0.0f;
  hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);
  printf("CPU execution time: %f milliseconds\n", cpu_time);
  
  // Save CPU results for verification
  float *P_cpu = static_cast<float *>(malloc(width * width * sizeof(float)));
  memcpy(P_cpu, P, width * width * sizeof(float));
  
  // Reset P for GPU computation
  memset(P, 0, width * width * sizeof(float));
  
  // GPU timing with CUDA events
  hipEvent_t start_gpu, stop_gpu, start_kernel, stop_kernel;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
  
  hipEventRecord(start_gpu);
  float *M_d, *N_d, *P_d;
  size_t size = width * width * sizeof(float);

  hipMalloc(reinterpret_cast<void **>(&M_d), size);
  hipMalloc(reinterpret_cast<void **>(&N_d), size);
  hipMalloc(reinterpret_cast<void **>(&P_d), size);

  hipMemcpy(M_d, M, size, hipMemcpyHostToDevice);
  hipMemcpy(N_d, N, size, hipMemcpyHostToDevice);

  const unsigned int numThreads = 512;
  unsigned int numBlocks = (width + numThreads - 1) / numThreads;

  hipEventRecord(start_kernel);
  gpu<<<numBlocks, numThreads>>>(M_d, N_d, P_d, width);
  hipEventRecord(stop_kernel);
  hipEventSynchronize(stop_kernel);
  
  float kernel_time = 0.0f;
  hipEventElapsedTime(&kernel_time, start_kernel, stop_kernel);
  printf("GPU kernel execution time: %f milliseconds\n", kernel_time);
  
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  hipMemcpy(P, P_d, size, hipMemcpyDeviceToHost);

  hipFree(M_d);
  hipFree(N_d);
  hipFree(P_d);

  hipEventRecord(stop_gpu);
  hipEventSynchronize(stop_gpu);
  
  float gpu_time = 0.0f;
  hipEventElapsedTime(&gpu_time, start_gpu, stop_gpu);
  printf("GPU execution time: %f milliseconds\n", gpu_time);
  
  // Verify results - calculate max error and mean error
  float max_error = 0.0f;
  float mean_error = 0.0f;
  int error_count = 0;
  
  for (int i = 0; i < width * width; ++i) {
    float error = fabs(P[i] - P_cpu[i]);
    if (error > max_error) {
      max_error = error;
    }
    if (error > 1e-3) { // Count significant errors
      error_count++;
    }
    mean_error += error;
  }
  mean_error /= (width * width);
  
  // Report verification results
  printf("Verification results:\n");
  printf("  Max error: %e\n", max_error);
  printf("  Mean error: %e\n", mean_error);
  printf("  Error count (>1e-3): %d out of %d elements\n", error_count, width * width);
  
  // Determine if verification passed using a reasonable threshold
  bool verification_passed = (max_error < 1e-2) && (error_count < width * width / 1000);
  printf("%s\n", verification_passed ? "Verification PASSED!" : "Verification FAILED!");
  
  // Calculate speedup
  printf("Speedup (CPU vs GPU kernel): %.2fx\n", cpu_time / kernel_time);
  printf("Speedup (CPU vs GPU total): %.2fx\n", cpu_time / gpu_time);
  
  // Cleanup events
  hipEventDestroy(start_cpu);
  hipEventDestroy(stop_cpu);
  hipEventDestroy(start_gpu);
  hipEventDestroy(stop_gpu);
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);

  // Free memory
  free(M);
  free(N);
  free(P);
  free(P_cpu);

  return 0;
}