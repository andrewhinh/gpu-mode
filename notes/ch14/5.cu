#include <chrono>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void jdsSpMV(const int *__restrict__ jds_row_ptr,
                        const int *__restrict__ col_idx,
                        const float *__restrict__ data,
                        const int *__restrict__ perm,
                        const float *__restrict__ x, float *__restrict__ y,
                        int num_rows, int max_row_nnz) {

  int row = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < num_rows) {
    int orig_row = perm[row];
    float sum = 0.0f;

    for (int j = 0; j < max_row_nnz; j++) {
      // Skip if this row has fewer non-zeros than the current jagged diagonal
      if (j >= jds_row_ptr[row + 1] - jds_row_ptr[row]) {
        break;
      }

      int idx = jds_row_ptr[row] + j;
      if (idx >= jds_row_ptr[row + 1]) {
        break;
      }

      sum += data[idx] * x[col_idx[idx]];
    }

    y[orig_row] = sum;
  }
}

// GPU implementation of JDS SpMV
void jdsSpMVHost(int *jds_row_ptr, int *col_idx, float *data, int *perm,
                 float *x, float *y, int num_rows, int num_cols,
                 int max_row_nnz) {

  int *d_jds_row_ptr, *d_col_idx, *d_perm;
  float *d_data, *d_x, *d_y;

  hipMalloc(&d_jds_row_ptr, (num_rows + 1) * sizeof(int));
  hipMalloc(&d_col_idx, jds_row_ptr[num_rows] * sizeof(int));
  hipMalloc(&d_data, jds_row_ptr[num_rows] * sizeof(float));
  hipMalloc(&d_perm, num_rows * sizeof(int));
  hipMalloc(&d_x, num_cols * sizeof(float));
  hipMalloc(&d_y, num_rows * sizeof(float));

  hipMemcpy(d_jds_row_ptr, jds_row_ptr, (num_rows + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_col_idx, col_idx, jds_row_ptr[num_rows] * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_data, data, jds_row_ptr[num_rows] * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_perm, perm, num_rows * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, num_cols * sizeof(float), hipMemcpyHostToDevice);

  hipMemset(d_y, 0, num_rows * sizeof(float));

  int blockSize = 256;
  int gridSize = (num_rows + blockSize - 1) / blockSize;
  jdsSpMV<<<gridSize, blockSize>>>(d_jds_row_ptr, d_col_idx, d_data, d_perm,
                                   d_x, d_y, num_rows, max_row_nnz);

  hipMemcpy(y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_jds_row_ptr);
  hipFree(d_col_idx);
  hipFree(d_data);
  hipFree(d_perm);
  hipFree(d_x);
  hipFree(d_y);
}

// CPU implementation of JDS SpMV for verification
void jdsSpMVHost_CPU(int *jds_row_ptr, int *col_idx, float *data, int *perm,
                     float *x, float *y, int num_rows, int num_cols,
                     int max_row_nnz) {
  // Initialize output vector to zero
  for (int i = 0; i < num_rows; i++) {
    y[i] = 0.0f;
  }

  // Process each JDS row
  for (int row = 0; row < num_rows; row++) {
    int orig_row = perm[row];
    float sum = 0.0f;

    for (int j = 0; j < max_row_nnz; j++) {
      // Skip if this row has fewer non-zeros than the current jagged diagonal
      if (j >= jds_row_ptr[row + 1] - jds_row_ptr[row]) {
        break;
      }

      int idx = jds_row_ptr[row] + j;
      if (idx >= jds_row_ptr[row + 1]) {
        break;
      }

      sum += data[idx] * x[col_idx[idx]];
    }

    y[orig_row] = sum;
  }
}

// Function to verify results
bool verifyResults(float *y_cpu, float *y_gpu, int num_rows,
                   float tolerance = 1e-5f) {
  for (int i = 0; i < num_rows; i++) {
    if (fabsf(y_cpu[i] - y_gpu[i]) > tolerance) {
      printf("Result mismatch at index %d: CPU = %f, GPU = %f\n", i, y_cpu[i],
             y_gpu[i]);
      return false;
    }
  }
  return true;
}

int main() {
  int num_rows = 4;
  int num_cols = 4;
  int max_row_nnz = 2;

  // Original matrix:
  // 1 0 7 0
  // 0 0 8 0
  // 0 4 3 0
  // 2 0 0 1

  int perm[4] = {0, 2, 3, 1}; // Rows in order of non-zero count
  int jds_row_ptr[5] = {0, 2, 4, 6,
                        7}; // Max 2 non-zeros per row, so num_rows+1 entries

  // Column indices and data arranged by jagged diagonals
  // First jagged diagonal (first non-zero in each row):
  // Row 0 (orig 0): col 0, val 1
  // Row 1 (orig 2): col 1, val 4
  // Row 2 (orig 3): col 0, val 2
  // Row 3 (orig 1): col 2, val 8

  // Second jagged diagonal (second non-zero in each row):
  // Row 0 (orig 0): col 2, val 7
  // Row 1 (orig 2): col 2, val 3
  // Row 2 (orig 3): col 3, val 1

  int col_idx[7] = {// First jagged diagonal
                    0, 1, 0, 2,
                    // Second jagged diagonal
                    2, 2, 3};

  float data[7] = {// First jagged diagonal
                   1.0f, 4.0f, 2.0f, 8.0f,
                   // Second jagged diagonal
                   7.0f, 3.0f, 1.0f};

  float x[4] = {1.0f, 2.0f, 3.0f, 4.0f};
  float y_gpu[4] = {0.0f, 0.0f, 0.0f, 0.0f};
  float y_cpu[4] = {0.0f, 0.0f, 0.0f, 0.0f};

  // Create CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Measure CPU implementation time
  auto cpu_start = std::chrono::high_resolution_clock::now();
  jdsSpMVHost_CPU(jds_row_ptr, col_idx, data, perm, x, y_cpu, num_rows,
                  num_cols, max_row_nnz);
  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<float, std::milli> cpu_duration = cpu_end - cpu_start;

  // Measure GPU implementation time
  hipEventRecord(start);
  jdsSpMVHost(jds_row_ptr, col_idx, data, perm, x, y_gpu, num_rows, num_cols,
              max_row_nnz);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float gpu_duration = 0.0f;
  hipEventElapsedTime(&gpu_duration, start, stop);

  // Verify results
  bool results_match = verifyResults(y_cpu, y_gpu, num_rows);

  // Print timing and verification results
  printf(
      "JDS SpMV Performance (%d rows, %d columns, %d max non-zeros per row):\n",
      num_rows, num_cols, max_row_nnz);
  printf("CPU Time: %.4f ms\n", cpu_duration.count());
  printf("GPU Time: %.4f ms\n", gpu_duration);
  printf("Speedup: %.2fx\n", cpu_duration.count() / gpu_duration);
  printf("Verification: %s\n\n", results_match ? "PASSED" : "FAILED");

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
